#include "hip/hip_runtime.h"

#include <thrust/adjacent_difference.h>
#include <thrust/extrema.h> // max_element
#include <thrust/iterator/zip_iterator.h>
#include <thrust/device_vector.h>

auto max_gap_device(thrust::device_vector<int> nums) -> int 
{
    auto diffs = thrust::device_vector<int>(nums.size());
    thrust::adjacent_difference(nums.begin(), nums.end(), diffs.begin());
    return *thrust::max_element(diffs.begin() + 1, diffs.end());
}

auto max_gap2_device(thrust::device_vector<int> nums) -> int 
{
    return thrust::transform_reduce(
        thrust::make_zip_iterator(thrust::make_tuple(nums.begin() + 1, nums.begin())), 
        thrust::make_zip_iterator(thrust::make_tuple(nums.end(),       nums.end() - 1)),
        [] __host__ __device__ (thrust::tuple<int, int> t) { return thrust::get<0>(t) - thrust::get<1>(t); }, 0, 
        [] __host__ __device__ (int a, int b) { return std::max(a, b); });
}   
